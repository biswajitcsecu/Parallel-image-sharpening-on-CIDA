
#define cimg_display 0

#include "CImg.h"
#include <hip/hip_runtime.h>
#include <npp.h>
#include <nppdefs.h>
#include <nppcore.h>
#include <nppi.h>
#include <npps.h>
#include <stdlib.h>
#include <iostream>


#define kNumCh 3

using namespace std;
using namespace cimg_library;

void sharpenNPP(Npp8u *arr, int w, int h) {
    //variables
    Npp8u *d_in = 0, *d_out = 0;
    Npp32f *d_filter = 0;
    
    const Npp32f filter[9] = {-1.0, -1.0, -1.0,
    -1.0, 9.0, -1.0,
    -1.0, -1.0, -1.0};
    
    //memory allocation
    hipMalloc(&d_out, kNumCh*w*h*sizeof(Npp8u));
    hipMalloc(&d_in, kNumCh*w*h*sizeof(Npp8u));
    hipMalloc(&d_filter, 9*sizeof(Npp32f));
    hipMemcpy(d_in, arr, kNumCh*w*h*sizeof(Npp8u),hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, 9*sizeof(Npp32f),hipMemcpyHostToDevice);
    
    //Parameters
    const NppiSize oKernelSize = {3, 3};
    const NppiPoint oAnchor = {1, 1};
    const NppiSize oSrcSize = {w, h};
    const NppiPoint oSrcOffset = {0, 0};
    const NppiSize oSizeROI = {w, h};
    
    //Filter
    nppiFilterBorder32f_8u_C3R(d_in, kNumCh*w*sizeof(Npp8u), oSrcSize, oSrcOffset, d_out, kNumCh*w*sizeof(Npp8u), oSizeROI, d_filter,
oKernelSize, oAnchor, NPP_BORDER_REPLICATE);    
    
    //from device to host
    hipMemcpy(arr, d_out, kNumCh*w*h*sizeof(Npp8u),  hipMemcpyDeviceToHost);
    
    //device deallocation
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_filter);
}



int main(int argc, char** argv) {
    try{
    //load image
    cimg_library::CImg<unsigned char> image("src.bmp");
    
    const int w = image.width();
    const int h = image.height();

    Npp8u *arr = (Npp8u*)malloc(kNumCh*w*h*sizeof(Npp8u));
    
    for (int r = 0; r < h; ++r) {
        for (int c = 0; c < w; ++c) {
            for (int ch = 0; ch < kNumCh; ++ch) {
                arr[kNumCh*(r*w + c) + ch] = image(c, r, ch);
            }
        }
    }

    // method call
    sharpenNPP(arr, w, h);
    
    for (int r = 0; r < h; ++r) {
        for (int c = 0; c < w; ++c) {
            for (int ch = 0; ch < kNumCh; ++ch) {
                image(c, r, ch) = arr[kNumCh*(r*w + c) + ch];
            }
        }
    }
    
    image.save_bmp("out.bmp");
    free(arr);
    }catch(std::exception &orb){
        std::cerr<<""<<orb.what()<<std::endl;
        return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;

}






